#include "hip/hip_runtime.h"
#include "distribution_evolution_tests.cuh"

double tol = 1.e-6;

SCENARIO("[DEVICE] Acceleration Update", "[d-acc]") {
    GIVEN("A thermal distribution of 5000 positions, help in a quadrupole trap with a Bz = 2.0") {
        int num_test = 5000;

        // Initialise trapping parameters
        trap_geo trap_parameters;
        trap_parameters.Bz = 2.0;
        trap_parameters.B0 = 0.;

        // Initialise rng
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_test*sizeof(hiprandState)));
        initialise_rng_states(num_test,
                              state,
                              false);

        // Initialise positions
        double3 *d_pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pos),
                                   num_test*sizeof(double3)));

        // Generate velocity distribution
        generate_thermal_positions(num_test,
                                   20.e-6,
                                   trap_parameters,
                                   state,
                                   d_pos);

        WHEN("The update_atom_accelerations function is called") {
            // Initialise accelerations
            double3 *d_test_acc;
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_test_acc),
                                       num_test*sizeof(double3)));

            // Generate accelerations
            update_accelerations(num_test,
                                 trap_parameters,
                                 d_pos,
                                 d_test_acc);;

            double3 *test_acc;
            test_acc = reinterpret_cast<double3*>(calloc(num_test,
                                                 sizeof(double3)));
            checkCudaErrors(hipMemcpy(test_acc,
                                       d_test_acc,
                                       num_test*sizeof(double3),
                                       hipMemcpyDeviceToHost));

            double mean_acc_x = mean_x(test_acc,
                                       num_test);
            double mean_acc_y = mean_y(test_acc,
                                       num_test);
            double mean_acc_z = mean_z(test_acc,
                                       num_test);

            double std_acc_x = std_dev_x(test_acc,
                                         num_test);
            double std_acc_y = std_dev_y(test_acc,
                                         num_test);
            double std_acc_z = std_dev_z(test_acc,
                                         num_test);

            THEN("The mean in each direction should be 0.") {
                REQUIRE(mean_acc_x <= 0. + std_acc_x / sqrt(num_test));
                REQUIRE(mean_acc_x >= 0. - std_acc_x / sqrt(num_test));
                REQUIRE(mean_acc_y <= 0. + std_acc_y / sqrt(num_test));
                REQUIRE(mean_acc_y >= 0. - std_acc_y / sqrt(num_test));
                REQUIRE(mean_acc_z <= 0. + std_acc_z / sqrt(num_test));
                REQUIRE(mean_acc_z >= 0. - std_acc_z / sqrt(num_test));
            }

            double expected_std_x_y = sqrt(trap_parameters.Bz*trap_parameters.Bz * gs*gs * muB*muB / 
                                           (48. * mass*mass));
            double expected_std_z = sqrt(trap_parameters.Bz*trap_parameters.Bz * gs*gs * muB*muB / 
                                           (12. * mass*mass));
            THEN("The standard deviation in each direction should be given by blah") {
                REQUIRE(std_acc_x <= expected_std_x_y + std_acc_x / sqrt(num_test));
                REQUIRE(std_acc_x >= expected_std_x_y - std_acc_x / sqrt(num_test));
                REQUIRE(std_acc_y <= expected_std_x_y + std_acc_y / sqrt(num_test));
                REQUIRE(std_acc_y >= expected_std_x_y - std_acc_y / sqrt(num_test));
                REQUIRE(std_acc_z <= expected_std_z + std_acc_z / sqrt(num_test));
                REQUIRE(std_acc_z >= expected_std_z - std_acc_z / sqrt(num_test));
            }

            hipFree(d_test_acc);
            free(test_acc);
        }

        hipFree(d_pos);
    }
}

SCENARIO("[DEVICE] Velocity Update", "[d-vel]") {
    GIVEN("A thermal distribution of 5000 positions, help in a quadrupole trap with a Bz = 2.0") {
        double init_T = 20.e-6;
        int num_test = 5000;

        // Initialise trapping parameters
        trap_geo trap_parameters;
        trap_parameters.Bz = 2.0;
        trap_parameters.B0 = 0.;

        // Initialise rng
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_test*sizeof(hiprandState)));

        initialise_rng_states(num_test,
                              state,
                              false);

        // Initialise positions
        double3 *d_pos;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pos),
                                   num_test*sizeof(double3)));

        // Generate velocity distribution
        generate_thermal_positions(num_test,
                                   20.e-6,
                                   trap_parameters,
                                   state,
                                   d_pos);

        // Initialise accelerations
        double3 *d_acc;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_acc),
                                   num_test*sizeof(double3)));

            // Generate accelerations
            update_accelerations(num_test,
                                 trap_parameters,
                                 d_pos,
                                 d_acc);

        WHEN("The update_velocities function is called with dt=1.e-6") {
            double dt = 1.e-6;
            // Initialise velocities
            double3 *d_test_vel;
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_test_vel),
                                       num_test*sizeof(double3)));

            // Generate velocity distribution
            generate_thermal_velocities(num_test,
                                        init_T,
                                        state,
                                        d_test_vel);

            double3 *test_vel;
            test_vel = reinterpret_cast<double3*>(calloc(num_test,
                                                  sizeof(double3)));
            checkCudaErrors(hipMemcpy(test_vel,
                                       d_test_vel,
                                       num_test*sizeof(double3),
                                       hipMemcpyDeviceToHost));

            double initial_kinetic_energy = mean_kinetic_energy(num_test,
                                                                test_vel);

            hipblasHandle_t cublas_handle;
            checkCudaErrors(hipblasCreate(&cublas_handle));
            update_velocities(num_test,
                              dt,
                              cublas_handle,
                              d_acc,
                              d_test_vel);
            hipblasDestroy(cublas_handle);

            checkCudaErrors(hipMemcpy(test_vel,
                                       d_test_vel,
                                       num_test*sizeof(double3),
                                       hipMemcpyDeviceToHost));

            double final_kinetic_energy = mean_kinetic_energy(num_test,
                                                              test_vel);

            THEN("The change in kinetic energy should be 0") {
                REQUIRE(final_kinetic_energy - initial_kinetic_energy > -tol);
                REQUIRE(final_kinetic_energy - initial_kinetic_energy < tol);
            }

            hipFree(d_test_vel);
            
        }

        hipFree(d_pos);
        hipFree(d_acc);
    }
}
