#include "random_number_generation_tests.cuh"

SCENARIO("[DEVICE] Uniform random number generation", "[d-urng]") {
    GIVEN("An appropriate seed") {
        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   sizeof(hiprandState)));
        initialise_rng_states(1,
                              state);

        WHEN("The random number generator is called") {
            double r;
            uniform_prng_launcher(1,
                                  state,
                                  &r);

            THEN("The result should be between 0 and 1") {
                REQUIRE(r >= 0.);
                REQUIRE(r <= 1.);
            }
        }

        // WHEN("We assign the local seed to the global seed") {
        //     g_rng = rng;
        //     unif01_Gen *gen;
        //     char* rng_name = "g_uniform_prng";
        //     gen = unif01_CreateExternGen01(rng_name,
        //                                    g_uniform_prng);

        //     THEN("We expect to pass small crush") {
        //         bbattery_SmallCrush(gen);
        //         bool complete = true;
        //         REQUIRE(complete);
        //     }

        //     unif01_DeleteExternGen01(gen);
        // }

        hipFree(state);
    }
}

SCENARIO("[DEVICE] Normally distributed random number generation", "[d-nrng]") {
    GIVEN("An array of appropriate seeds") {
        int num_test = 10000;

        hiprandState *state;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&state),
                                   num_test*sizeof(hiprandState)));
        initialise_rng_states(num_test,
                              state);

        WHEN("We generate 10,000 numbers using a mean of 0 and a std of 1") {
            double *d_test_values;
            checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_test_values),
                                       num_test*sizeof(double)));
            gaussian_prng(num_test,
                          state,
                          d_test_values);

            double *test_values;
            test_values = reinterpret_cast<double*>(calloc(num_test,
                                                    sizeof(double)));
            checkCudaErrors(hipMemcpy(test_values,
                                       d_test_values,
                                       num_test*sizeof(double),
                                       hipMemcpyDeviceToHost));

            THEN("The result should pass the back-of-the-envelope test") {
                double val_mean = mean(test_values,
                                       num_test);
                double val_std  = std_dev(test_values,
                                          num_test);
                double val_max = *std::max_element(test_values,
                                                   test_values+num_test);
                double val_min = *std::min_element(test_values,
                                                   test_values+num_test);

                double Z_max = z_score(val_max,
                                       val_mean,
                                       val_std);
                double Z_min = z_score(val_min,
                                       val_mean,
                                       val_std);
                REQUIRE(Z_max <= 4.);
                REQUIRE(Z_min >=-4.);
            }

            hipFree(d_test_values);
            free(test_values);
            // Also need to implement a more rigorous test
        }

        hipFree(state);
    }
}
